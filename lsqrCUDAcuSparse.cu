#include "hip/hip_runtime.h"
#include "lsqrCUDAcuSparse.h"

CPUMatrix cusparseLSQR(const CPUMatrix &A, const CPUMatrix &b, double ebs){
    hipsparseStatus_t status;
    hipsparseHandle_t handle;
    status = hipsparseCreate(&handle);
    cuSPARSECheck(status,__LINE__);
    GPUMatrix u = matrix_alloc_gpu(b.height,b.width);
    GPUMatrix v = matrix_alloc_gpu(b.height,b.width);
    GPUMatrix w = matrix_alloc_gpu(b.height,b.width);
    GPUMatrix x = matrix_alloc_gpu(b.height,b.width);
    GPUMatrix GPUb = matrix_alloc_gpu(b.height,b.width);
    GPUMatrix tempVector = matrix_alloc_gpu(b.height,b.width); 
    matrix_upload(b,GPUb);
    CPUMatrix res = cusparseLSQR_aux(A,GPUb,u,v,w,x,tempVector,ebs);
    return res; 
}

CPUMatrix cusparseLSQR_aux(const CPUMatrix &A, const GPUMatrix &VECb,GPUMatrix &VECu,GPUMatrix &VECv,GPUMatrix &VECw,GPUMatrix &VECx,GPUMatrix &tempVector,double ebs){
    double beta, alpha, phi, phi_tag, rho, rho_tag, c, s, theta, tempDouble, tempDouble2,curr_err,prev_err,improvment;
    size_t tempInt;
    double *buffer;
    hipsparseStatus_t status;
    hipsparseHandle_t handle;
    status = hipsparseCreate(&handle);
    cuSPARSECheck(status,__LINE__);
    prev_err = 100000000; 
    hipsparseSpMatDescr_t spMatrixA;
    hipsparseDnVecDescr_t b,u,v,w,x,tempDense;
    status = hipsparseCreateCsr(&spMatrixA,A.height,A.width,A.elementSize,A.csrRow,A.csrCol,A.elements,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
    hipsparseCreateDnVec(&b,VECb.height,VECb.elements,HIP_R_64F);
    hipsparseCreateDnVec(&u,VECb.height,VECu.elements,HIP_R_64F);
    hipsparseCreateDnVec(&v,VECb.height,VECv.elements,HIP_R_64F);
    hipsparseCreateDnVec(&w,VECb.height,VECw.elements,HIP_R_64F);
    hipsparseCreateDnVec(&x,VECb.height,VECx.elements,HIP_R_64F);
    hipsparseCreateDnVec(&tempDense,VECb.height,tempVector.elements,HIP_R_64F);

	//init stage
    //beta = norm(b)
    beta = normalVectorNorm(b,tempVector);
    beta =2;//TODO!!
    //u = b/beta
    copyVector(u,b,tempVector);
    printDenseVector(u,"u",tempVector);
    scaleNormalvector(u,1/beta,tempVector);
    printDenseVector(u,"u2",tempVector);
    //v = A'*u
    tempDouble = 1; tempDouble2 = 0;
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE,&tempDouble,spMatrixA,u,&tempDouble2,v,HIP_R_64F,HIPSPARSE_CSRMV_ALG1,&tempInt);
    hipMalloc(&buffer, tempInt);
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE,&tempDouble,spMatrixA,u,&tempDouble2,v,HIP_R_64F,HIPSPARSE_CSRMV_ALG1,&buffer);
    //alpha = norm(v)
    alpha = normalVectorNorm(v,tempVector);
    //v = v/alpha;
    scaleNormalvector(v,1/alpha,tempVector);
    //w = v;
    copyVector(w,v,tempVector);
    phi_tag = beta; rho_tag = alpha;
	int i = 0, counter = 0;
	while(true){
		//next bidiagonlization
        // u = A * v - alpha * u;
        tempDouble = 1; tempDouble2 = (-1)*alpha;
        hipsparseSpMV(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,&tempDouble,spMatrixA,v,&tempDouble2,u,HIP_R_64F,HIPSPARSE_CSRMV_ALG1,&buffer);
        //beta = norm(u);
        beta = normalVectorNorm(u,tempVector);
        // u = u / beta;
        scaleNormalvector(u,beta,tempVector);
        // v = A' * u - beta * v;
        tempDouble = 1; tempDouble2 = (-1)*beta;
        hipsparseSpMV(handle,HIPSPARSE_OPERATION_TRANSPOSE,&tempDouble,spMatrixA,u,&tempDouble2,v,HIP_R_64F,HIPSPARSE_CSRMV_ALG1,&buffer);
        //alpha = norm(v)
        alpha = normalVectorNorm(v,tempVector);
        //v = v/alpha;
        scaleNormalvector(v,1/alpha,tempVector);
		//next orthogonal transformation
		rho = sqrt(pow (rho_tag, 2.0) + pow (beta, 2.0));
		c = rho_tag / rho;
		s = beta / rho;
		theta = s * alpha;
		rho_tag = (-1) * c * alpha;
		phi = c * phi_tag;
		phi_tag = s * phi_tag;
		//printf("constants: alpha: %.6f beta:%.6f\n",alpha,beta);
		//printf("constants: rho: %.6f c: %.6f s: %.6f theta: %.6f rho_tag: %.6f phi: %.6f\n phi_tag: %.6f\n",rho,c,s,theta,rho_tag,phi,phi_tag);
        //updating x,w
        copyVector(tempDense,w,tempVector);
        scaleNormalvector(tempDense,phi/rho,tempVector); 
        //x = x + (phi / rho) * w ;          
        vectorAddSub(x,tempDense,true,tempVector);
        //	w = -(theta / rho) * w + v;
        scaleNormalvector(w,(theta/rho)*(-1),tempVector); 
        vectorAddSub(w,v,true,tempVector);
        //check for convergence
        tempDouble = 1; tempDouble2 = (-1);
        copyVector(tempDense,b,tempVector);
        hipsparseSpMV(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,&tempDouble,spMatrixA,x,&tempDouble2,tempDense,HIP_R_64F,HIPSPARSE_CSRMV_ALG1,&buffer);
		//residual = norm(A*x - b);
        //Ax - b (result in tempDense)
        curr_err = normalVectorNorm(tempDense,tempVector);
        improvment = prev_err-curr_err;
        printf("line: %d size of error: %.6f improvment of: %.6f\n",i,curr_err,improvment);i++;
        if(i==A.height) break;
    }
    CPUMatrix result = matrix_alloc_cpu(VECb.height,VECb.width);
    hipsparseDnVecGetValues(x,(void**)&tempVector.elements);
    matrix_download(tempVector,result);
    cusparseClean(handle,spMatrixA);
	return result;
}

void cusparseClean(hipsparseHandle_t handle, hipsparseSpMatDescr_t &A){
    hipsparseStatus_t status;
    status = hipsparseDestroySpMat(A);
    status = hipsparseDestroy(handle);
    cuSPARSECheck(status,__LINE__);
}


void cuSPARSECheck(hipsparseStatus_t status, int line){
	if(status != HIPSPARSE_STATUS_SUCCESS){
		printf("error code %d, line(%d)\n", status, line);
		exit(EXIT_FAILURE);
	}
}

double normalVectorNorm(hipsparseDnVecDescr_t src, GPUMatrix temp){
    hipsparseDnVecGetValues(src,(void**)&temp.elements);
    return getNorm2(temp);
}
void scaleNormalvector(hipsparseDnVecDescr_t src,double alpha,GPUMatrix temp){
    hipsparseDnVecGetValues(src,(void**)&temp.elements);
    printNormalVector(temp, "temp in scale");
    GPUMatrix res = multiply_scalar_vector(temp,alpha);
    printNormalVector(res, "Res in scale");
    hipsparseDnVecSetValues(src,res.elements);
}
void vectorAddSub(hipsparseDnVecDescr_t a, hipsparseDnVecDescr_t b, bool sign,GPUMatrix temp){  // result overrides to a
    GPUMatrix temp2 = matrix_alloc_gpu(temp.height,temp.width); 
    hipsparseDnVecGetValues(a,(void**)&temp.elements);
    hipsparseDnVecGetValues(b,(void**)&temp2.elements);   
    GPUMatrix res = get_add_subtract_vector(temp,temp2,sign);
    hipsparseDnVecSetValues(a,res.elements);
}
void copyVector(hipsparseDnVecDescr_t dst,hipsparseDnVecDescr_t src,GPUMatrix temp){
    hipsparseDnVecGetValues(src,(void**)&temp.elements);
    hipsparseDnVecSetValues(dst,temp.elements);
}

void printDenseVector(hipsparseDnVecDescr_t src,const char* name,GPUMatrix temp){
    hipsparseDnVecGetValues(src,(void**)&temp.elements);
    printf("%s: ",name);
	CPUMatrix tempCPUMatrix = matrix_alloc_cpu(temp.height,temp.width);
	matrix_download(temp,tempCPUMatrix);
	for(int i = 0; i < tempCPUMatrix.height; i++){
		printf("%lf ", tempCPUMatrix.elements[i]);
	}
	printf("\n");
}
void printNormalVector(GPUMatrix x, const char* name){
	printf("%s: ",name);
	CPUMatrix tempCPUMatrix = matrix_alloc_cpu(x.height,x.width);
	matrix_download(x,tempCPUMatrix);
	for(int i = 0; i < tempCPUMatrix.height; i++){
		printf("%lf ", tempCPUMatrix.elements[i]);
	}
	printf("\n");
}