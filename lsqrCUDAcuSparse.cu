#include "hip/hip_runtime.h"
#include "lsqrCUDAcuSparse.h"
#include "lsqr.h"
#include "matrix.h"
#include <hipsparse.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
CPUMatrix sparseLSQR(const CPUMatrix &A, const CPUMatrix &b, double ebs){
    hipsparseStatus_t status;
    hipsparseHandle_t handle;
    status = hipsparseCreate(&handle);
    cuSPARSECheck(status,__line);
    hipsparseSpMatDescr_t spMatrixA;
    hipsparseSpVecDescr_t spVectorb;
    hipsparseDnVecDescr_t u,v,w,x,tempVector;

    hipsparseCreateCsr(&A,A.height,A.rows,A.elementSize,A.csrRow,A.csrCol,A.elements,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
    hipsparseCreateSpVec(&spVectorb,b.height,b.elementSize,b.csrCol,b.elements,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
    
    double *zeros = new double[b.elementSize]; //to init helper vectors
    
    hipsparseCreateDnVec(&u,b.height,b.elementSize,b.csrCol,zeros,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
    hipsparseCreateDnVec(&v,b.height,b.elementSize,b.csrCol,zeros,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
    hipsparseCreateDnVec(&w,b.height,b.elementSize,b.csrCol,zeros,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
    hipsparseCreateDnVec(&x,b.height,b.elementSize,b.csrCol,zeros,HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
    
    hipsparseDestroy(handle);
    return sparseLSQR_aux();
}

CPUMatrix sparseLSQR_aux(const hipsparseSpMatDescr_t &A, const hipsparseSpVecDescr_t &b,GPUMatrix &u,GPUMatrix &v,GPUMatrix &w,GPUMatrix &x,GPUMatrix &tempVector,double ebs){
    double beta, alpha, phi, phi_tag, rho, rho_tag, c, s, theta, tempDouble, tempDouble2,curr_err,prev_err,improvment;
    hipsparseStatus_t status;
    hipsparseHandle_t handle;
    status = hipsparseCreate(&handle);
    cuSPARSECheck(status,__line)
	prev_err = 100000000; 
	//init stage
	//beta = norm(b)
	//u = b/beta
	//v = A'*u
	//alpha = norm(v)
	//v = v/alpha;
    //w = v;
    //phi_hat = beta;
    //rho_hat = alpha;

	int i = 0, counter = 0;
	while(true){
		//next bidiagonlization
		// u = A * v - alpha * u;
		//beta = norm(u);
		// u = u / beta;
		// v = A' * u - beta * v;
		//alpha = norm(v)
		//v = v/alpha;
		//next orthogonal transformation
		rho = sqrt(pow (rho_tag, 2.0) + pow (beta, 2.0));
		c = rho_tag / rho;
		s = beta / rho;
		theta = s * alpha;
		rho_tag = (-1) * c * alpha;
		phi = c * phi_tag;
		phi_tag = s * phi_tag;
		//printf("constants: alpha: %.6f beta:%.6f\n",alpha,beta);
		//printf("constants: rho: %.6f c: %.6f s: %.6f theta: %.6f rho_tag: %.6f phi: %.6f\n phi_tag: %.6f\n",rho,c,s,theta,rho_tag,phi,phi_tag);
		//updating x,w
		//x =  (phi / rho) * w + x;             (in cublas : x is y, w is x)
		//	w = v - (theta / rho) * w ;
		//check for convergence
		//residual = norm(A*x - b);
        //Ax - b (result in tempVector)
    }

}

void cuSPARSECheck(hipsparseStatus_t status, int line){
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("error code %d, line(%d)\n", status, line);
		exit(EXIT_FAILURE);
	}
}

double sparseVectorNorm(hipsparseSpVecDescr_t vector){
    return 0.0;
}
void scaleVector(hipsparseSpVecDescr_t vector){

}


/*
    hipsparseAxpby() - for vector addition
    cusparseCsrmvEx() matrix-vector multiplication
    cublas norm

    A is an m×n sparse matrix that is defined in CSR storage format by the three arrays csrValA, csrRowPtrA, and csrColIndA); x and y are vectors;

*/