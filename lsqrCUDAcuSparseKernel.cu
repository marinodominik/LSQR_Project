#include "hip/hip_runtime.h"
#include "lsqrCUDAcuSparseKernel.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include ""


#define BLOCK_SIZE 32            //max threads in a block

__global__ void sqaure_vector(const double *vector, double *tmp, const int size);
__global__ void norm2(const double *in_data, double *result, int size);
__global__ void add_subtract_vector(double *a, const double *b, const bool operation, const int size);  
__global__ void scalar_vector(double *in_data, const double scalar, const int size);
__global__ void matrix_vector_multiplication(const int n_row, const GPUMatrix &A_sparse, const GPUMatrix &vector_dense, GPUMatrix result);



inline unsigned int div_up(unsigned int numerator, unsigned int denominator) { //numerator = zähler, denumerator = nenner
	unsigned int result = numerator / denominator;
	if (numerator % denominator) ++result;
	return result;
}



/*
<<<<<<<<<<-------------------- NORM ----------------------------->>>>>>>>>>>>>>
*/

__global__ void sqaure_vector(const double *vector, double *tmp, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= size) { 
        return;
    } else {
        tmp[i] = vector[i] * vector[i];
    }

    __syncthreads();
}


__global__ void norm2(const double *in_data, double *result,int size) {
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid < size){
        sdata[tid] = in_data[i];        //load global data in sh_memory
    }else{
        sdata[tid] = 0; 
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    //thread 0 writes in result back to global memory
    if (tid == 0) {
        result[blockIdx.x] = sdata[0]; //Da wir n-grids haben, werden die zahlen für jeden block in eine eigene zelle im global gespeichert
    }
}


double getNorm2(const GPUMatrix denseVector) {
    GPUMatrix tmp = matrix_alloc_gpu(denseVector.height, denseVector.width);

    int grids = div_up(denseVector.height, BLOCK_SIZE * BLOCK_SIZE);
    
    double *result = new double[grids];
    hipMalloc(&result, grids * sizeof(double));
    
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    int sh_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(double);
    
    sqaure_vector<<<grids, dimBlock>>>(denseVector.elements, tmp.elements, tmp.height * tmp.width); 
    norm2<<<grids, dimBlock, sh_memory_size>>>(tmp.elements, result);
    
    double *values = new double[grids]; 
    hipMemcpy(values, result, grids * sizeof(double), hipMemcpyDeviceToHost);
    
    double norm = 0.0;
    for (int i= 0; i< grids; i++) {
        norm += values[i];
    }

    matrix_free_gpu(tmp);
    delete[] values;

    return sqrt(norm);
}


/*
<<<<<<<<<<-------------------- END NORM ----------------------------->>>>>>>>>>>>>>>>>
*/





/*
<<<<<<<<<<-------------------- ADDITION AND SUBSTRACTION ----------------------------->>>>>>>>>>>>>>>>>
*/

void get_add_subtract_vector(GPUMatrix denseA, const GPUMatrix denseB, const bool operation) {
    int grids = div_up(denseA.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);

    add_subtract_vector<<<grids, dimBlock>>>(denseA.elements, denseB.elements, operation, denseA.width * denseA.height);
}



__global__ void add_subtract_vector(double *a, const double *b, const bool operation, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    //check if index out of range of vector
    if(i >= size) return;

    if(operation == true) {
        a[i] = a[i] + b[i];

    } else {
        a[i] = a[i] - b[i];
    }
    __syncthreads();
}


/*
<<<<<<<<<<-------------------- END ADDITON AND SUBSTRACTION ----------------------------->>>>>>>>>>>>>>>>>
*/




/*
<<<<<<<<<<-------------------- MULTIPLY SCALAR ----------------------------->>>>>>>>>>>>>>>>>
*/

void multiply_scalar_vector(GPUMatrix vector, const double scalar) {
    int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);

    scalar_vector<<<grids, dimBlock>>>(vector.elements, scalar, vector.height * vector.width);
}


__global__ void scalar_vector(double *in_data, const double scalar, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        in_data[i] = scalar * in_data[i];
    }
    __syncthreads();
}
/*
<<<<<<<<<<-------------------- END MULTIPLICATION SCALAR ----------------------------->>>>>>>>>>>>>>>>>
*/




/*
<<<<<<<<<<-------------------- CSR MATRIX MULTIPLY WITH DENSE VECTOR ----------------------------->>>>>>>>>>>>>>>>>
*/

__global__ void matrix_vector_multiplication(const int n_rows, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n_rows) {
        const int row_start = rowPtr[row];
        const int row_end = rowPtr[row + 1];

        double sum = 0.0;
        for (int idx = row_start; idx < row_end; idx++) {
            int col = colIdx[idx];
            sum += elements[idx] * x[col];
        }
        printf("sum: %lf\n", sum);
        result[row] = sum;
    }
    __syncthreads();
}


GPUMatrix get_csr_matrix_vector_multiplication(const GPUMatrix matrix, const GPUMatrix vector) {
    GPUMatrix result = matrix_alloc_gpu(vector.height, vector.width);

    int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    //int sh_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(double);
    matrix_vector_multiplication<<<grids, dimBlock>>>(matrix.height, matrix.elements, matrix.csrRow, matrix.csrCol, vector.elements, result.elements);

    return result;
}


/*
<<<<<<<<<<-------------------- END MATRIX VECTOR MULTIPLICATION ----------------------------->>>>>>>>>>>>>>>>>
*/




GPUMatrix lsqr_algrithm(const GPUMatrix &A, const GPUMatrix &b, const double lambda, const double ebs) {
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    cuSPARSECheck(__LINE__);

    GPUMatrix A_transpose = matrix_alloc_sparse_gpu(A.height, A.width, A.elementSize, A.rowSize, A.columnSize);
    hipMemcpy (A_transpose.elements, A.elements, A.elementSize * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy (A_transpose.csrRow, A.csrRow, A.rowSize * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy (A_transpose.csrCol, A.csrCol, A.columnSize * sizeof(int), hipMemcpyDeviceToDevice);

    cuSPARSECheck(__LINE__);
    hipsparseCsr2cscEx2();
    cuSPARSECheck(__LINE__);

    
    GPUMatrix x = matrix_alloc_gpu(b.height, b.width);
    GPUMatrix w = matrix_alloc_gpu(b.height, b.width);
    GPUMatrix u = matrix_alloc_gpu(b.height, b.width);
    hipMemcpy (u.elements, b.elements, b.height*sizeof(double), hipMemcpyDeviceToDevice);
    GPUMatrix v = matrix_alloc_gpu(b.height, b.width);

    /* INIZALIZATION PART */
    //beta = norm(b);
    double beta = getNorm2(u);

    //u = b/beta;
    multiply_scalar_vector(u, beta);

    GPUMatrix v = get_csr_matrix_vector_multiplication(A_transpose, u);

    v = A'*u;
    alpha = norm(v);
    v = v/alpha;
    w = v;
    x = 0;
    phi_hat = beta;
    rho_hat = alpha;
    % (2) iterate
    it_max = 10;
    epsilon = 10^-3;
    history = zeros(length(b),0);
    history(:,end+1) = x;

    return b; 
}



CPUMatrix sparseLSQR_with_kernels(const CPUMatrix &A, const CPUMatrix &b, const double lambda, const double ebs) {
    CPUMatrix resultCPU = matrix_alloc_cpu(b.height, b.width);
    GPUMatrix resultGPU = matrix_alloc_gpu(b.height, b.width);

    GPUMatrix A_gpu = matrix_alloc_sparse_gpu(A.height, A.width, A.elementSize, A.rowSize, A.columnSize);
    GPUMatrix b_gpu = matrix_alloc_gpu(b.height, b.width);
    
    /* upload Matrix, vector */
    matrix_upload_cuSparse(A, A_gpu);
    matrix_upload(b, b_gpu);

    resultGPU = lsqr_algrithm(A_gpu, b_gpu, lambda, ebs);

    //printVector(b.height * b.width, resultGPU, "add vector");

    /* Download result */
    matrix_download(resultGPU, resultCPU);

    /* free GPU memory */
    hipFree(resultGPU.elements);
    hipFree(A_gpu.elements);
    hipFree(b_gpu.elements);

    return resultCPU;
}





/*
    beta = norm(b);
u = b/beta;
v = A'*u;
alpha = norm(v);
v = v/alpha;
w = v;
x = 0;
phi_hat = beta;
rho_hat = alpha;
% (2) iterate
it_max = 10;
epsilon = 10^-3;
history = zeros(length(b),0);
history(:,end+1) = x;
for i = 1:it_max
    % (3) bidiagonalization
    u = A * v - alpha * u;
    beta = norm(u);
    u = u / beta;
    v = A' * u - beta * v;
    alpha = norm(v);
    v = v / alpha;
    % (4) orthogonal transformation
    rho = sqrt(rho_hat^2 + beta^2);
    c = rho_hat / rho;
    s = beta / rho;
    theta = s * alpha;
    rho_hat = -c * alpha;
    phi = c * phi_hat;
    phi_hat = s * phi_hat;
    % (5) update x, w
    x = x + (phi / rho) * w;
    w = v - (theta / rho) * w;
    history(:,end+1) = x;
    residual = norm(A*x - b);
    if(residual < epsilon)
        disp(['terminated after ',num2str(i),' iterations'])
        disp(['final residual: ',num2str(residual)])
        return
    end
end
*/