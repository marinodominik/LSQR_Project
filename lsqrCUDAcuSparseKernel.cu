#include "hip/hip_runtime.h"
#include "lsqrCUDAcuSparseKernel.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include ""
#include <hipsparse.h>

#define BLOCK_SIZE 32            //max threads in a block

__global__ void sqaure_vector(const double *vector, double *tmp, const int size);
__global__ void norm2(const double *in_data, double *result, int size);
__global__ void add_subtract_vector(double *a, double *b, bool operation, int size);  
__global__ void scalar_vector(double *in_data, const double scalar, const int size);
__global__ void matrix_vector_multiplication(const int n_rows, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result);
__global__ void matrix_vector_multiplication_sh(const int n_row, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result);


inline unsigned int div_up(unsigned int numerator, unsigned int denominator) { //numerator = zähler, denumerator = nenner
	unsigned int result = numerator / denominator;
	if (numerator % denominator) ++result;
	return result;
}


GPUMatrix transpose_matrix(GPUMatrix A) {
    GPUMatrix A_transpose = matrix_alloc_sparse_gpu(A.height, A.width, A.elementSize, A.rowSize, A.columnSize);
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    size_t tempInt;
    double *buffer;

    hipsparseCsr2cscEx2_bufferSize(handle, A.height, A.width, A.elementSize,
                                  A.elements, A.csrRow, A.csrCol,
                                  A_transpose.elements, A_transpose.csrCol,A_transpose.csrRow, 
                                  HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &tempInt);

    hipMalloc(&buffer, tempInt);

    hipsparseCsr2cscEx2(handle, A.height, A.width, A.elementSize,
                       A.elements, A.csrRow, A.csrCol,
                       A_transpose.elements, A_transpose.csrRow, A_transpose.csrCol, 
                       HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);
    
    return A_transpose;
}



/*
<<<<<<<<<<-------------------- NORM ----------------------------->>>>>>>>>>>>>>
*/

__global__ void sqaure_vector(const double *vector, double *tmp, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= size) { 
        return;
    } else {
        tmp[i] = vector[i] * vector[i];
    }

    __syncthreads();
}


__global__ void norm2(const double *in_data, double *result,int size) {
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid < size){
        sdata[tid] = in_data[i];        //load global data in sh_memory
    }else{
        sdata[tid] = 0; 
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    //thread 0 writes in result back to global memory
    if (tid == 0) {
        result[blockIdx.x] = sdata[0]; //Da wir n-grids haben, werden die zahlen für jeden block in eine eigene zelle im global gespeichert
    }
}


double getNorm2(const GPUMatrix denseVector) {
    GPUMatrix tmp = matrix_alloc_gpu(denseVector.height, denseVector.width);

    int grids = div_up(denseVector.height, BLOCK_SIZE * BLOCK_SIZE);

    double *result;
    hipMalloc(&result, grids * sizeof(double));
    
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    int sh_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(double);
    
    sqaure_vector<<<grids, dimBlock>>>(denseVector.elements, tmp.elements, tmp.height * tmp.width); 
    norm2<<<grids, dimBlock, sh_memory_size>>>(tmp.elements, result, tmp.height * tmp.width);

    
    double *values = new double[grids]; 
    hipMemcpy(values, result, grids * sizeof(double), hipMemcpyDeviceToHost);

    double norm = 0.0;
    for (int i= 0; i< grids; i++) {
        norm += values[i];
    }

    matrix_free_gpu(tmp);
    delete[] values;
    hipFree(result);

    return sqrt(norm);
}


/*
<<<<<<<<<<-------------------- END NORM ----------------------------->>>>>>>>>>>>>>>>>
*/





/*
<<<<<<<<<<-------------------- ADDITION AND SUBSTRACTION ----------------------------->>>>>>>>>>>>>>>>>
*/

void get_add_subtract_vector(GPUMatrix denseA, GPUMatrix denseB, bool operation) {
    printf("get add\n");
    int grids = div_up(denseA.height, BLOCK_SIZE * BLOCK_SIZE);
    kernelCheck(__LINE__);
    printf("%d\n", grids);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    kernelCheck(__LINE__);
    printf("before");
    kernelCheck(__LINE__);
    add_subtract_vector<<<grids, dimBlock>>>(denseA.elements, denseB.elements, operation, denseA.width * denseA.height);
    kernelCheck(__LINE__);
    printf("after");
}



__global__ void add_subtract_vector(double *a, double *b, bool operation, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    //check if index out of range of vector
    if(i >= size) return;

    if(operation == true) {
        a[i] = a[i] + b[i];

    } else {
        a[i] = a[i] - b[i];
    }
    __syncthreads();
}


/*
<<<<<<<<<<-------------------- END ADDITON AND SUBSTRACTION ----------------------------->>>>>>>>>>>>>>>>>
*/




/*
<<<<<<<<<<-------------------- MULTIPLY SCALAR ----------------------------->>>>>>>>>>>>>>>>>
*/

void multiply_scalar_vector(GPUMatrix vector, const double scalar) {
    int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);

    scalar_vector<<<grids, dimBlock>>>(vector.elements, scalar, vector.height * vector.width);
}


__global__ void scalar_vector(double *in_data, const double scalar, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        in_data[i] = scalar * in_data[i];
    }
    __syncthreads();
}
/*
<<<<<<<<<<-------------------- END MULTIPLICATION SCALAR ----------------------------->>>>>>>>>>>>>>>>>
*/




/*
<<<<<<<<<<-------------------- CSR MATRIX MULTIPLY WITH DENSE VECTOR ----------------------------->>>>>>>>>>>>>>>>>
*/

__global__ void matrix_vector_multiplication(const int n_rows, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n_rows) {
        const int row_start = rowPtr[row];
        const int row_end = rowPtr[row + 1];

        double sum = 0.0;
        for (int idx = row_start; idx < row_end; idx++) {
            int col = colIdx[idx];
            sum += elements[idx] * x[col];
        }
        result[row] = sum;
    }
    __syncthreads();
}

__global__ void matrix_vector_multiplication_sh(const int n_row, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result) {
    
}


GPUMatrix get_csr_matrix_vector_multiplication_sh(const GPUMatrix matrix, const GPUMatrix vector) {
    GPUMatrix result = matrix_alloc_gpu(vector.height, vector.width);

    int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    int sh_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(double);

    matrix_vector_multiplication_sh<<<grids, dimBlock, sh_memory_size>>>(matrix.height, matrix.elements, matrix.csrRow, matrix.csrCol, vector.elements, result.elements);

    return result;
}



GPUMatrix get_csr_matrix_vector_multiplication(const GPUMatrix matrix, const GPUMatrix vector) {
    GPUMatrix result = matrix_alloc_gpu(vector.height, vector.width);

    int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    matrix_vector_multiplication<<<grids, dimBlock>>>(matrix.height, matrix.elements, matrix.csrRow, matrix.csrCol, vector.elements, result.elements);

    return result;
}


/*
<<<<<<<<<<-------------------- END MATRIX VECTOR MULTIPLICATION ----------------------------->>>>>>>>>>>>>>>>>
*/


GPUMatrix lsqr_algrithm(const GPUMatrix &A, const GPUMatrix &b, const double lambda, const double ebs) {
    GPUMatrix x = matrix_alloc_gpu(b.height, b.width);
    printValuesKernel(A, "A");

    GPUMatrix A_transpose = transpose_matrix(A);

    printValuesKernel(A_transpose, "A_transpose");

    return x; 
}


CPUMatrix sparseLSQR_with_kernels(const CPUMatrix &A, const CPUMatrix &b, const double lambda, const double ebs) {
    CPUMatrix resultCPU = matrix_alloc_cpu(b.height, b.width);
    GPUMatrix resultGPU = matrix_alloc_gpu(b.height, b.width);

    GPUMatrix A_gpu = matrix_alloc_sparse_gpu(A.height, A.width, A.elementSize, A.rowSize, A.columnSize);
    GPUMatrix b_gpu = matrix_alloc_gpu(b.height, b.width);

    /* upload Matrix, vector */
    matrix_upload_cuSparse(A, A_gpu);
    matrix_upload(b, b_gpu);
    
    printf("hier1\n");
    resultGPU = lsqr_algrithm(A_gpu, b_gpu, lambda, ebs);
    printf("hier2");

    /* Download result */
    matrix_download(resultGPU, resultCPU);

    /* free GPU memory */
    matrix_free_sparse_gpu(A_gpu);
    matrix_free_gpu(b_gpu);
    matrix_free_gpu(resultGPU);

    return resultCPU;
}


void printVectorKernel(int iteration,GPUMatrix x, const char* name){
	printf("%s: ",name);
	CPUMatrix tempCPUMatrix = matrix_alloc_cpu(x.height, x.width);
	matrix_download(x ,tempCPUMatrix);
	//printf("iteration number: %d\n", iteration);
	for(int i = 0; i < 9; i++){
		printf("%lf ", tempCPUMatrix.elements[i]);
	}
	printf("\n");
}

void printValuesKernel(GPUMatrix x, const char *name) {
    printf("%s: ",name);
	CPUMatrix tempCPUMatrix = matrix_alloc_sparse_cpu(x.height, x.width, x.elementSize, x.rowSize, x.columnSize);
    matrix_download_cuSparse(x ,tempCPUMatrix);
    

    for(int i = 0; i < 9; i++){
		printf("%lf ", tempCPUMatrix.elements[i]);
    }
    printf("\n Row:");

    for(int i = 0; i < 4; i++){
		printf("%d ", tempCPUMatrix.csrRow[i]);
    }
    printf("\n Col:");
    for(int i = 0; i < 9; i++){
		printf("%d ", tempCPUMatrix.csrCol[i]);
    }
    printf("\n");
}


void kernelCheck(int line){
	const hipError_t err = hipGetLastError();                            
	if (err != hipSuccess) {                                              
    	const char *const err_str = hipGetErrorString(err);               
    	std::cerr << "Cuda error in " << __FILE__ << ":" << line - 1   
            << ": " << err_str << " (" << err << ")" << std::endl;   
            exit(EXIT_FAILURE);                                                                    
	}
}