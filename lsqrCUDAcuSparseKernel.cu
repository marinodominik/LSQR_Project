#include "hip/hip_runtime.h"
#include "lsqrCUDAcuSparseKernel.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include ""


#define BLOCK_SIZE 32            //max threads in a block

__global__ void sqaure_vector(const double *vector, double *result, const int size);
__global__ void norm2(const double *in_data, double *result);
__global__ void add_subtract_vector(const double *a, const double *b, double *c, const bool operation, const int size);
__global__ void scalar_vector(const double *in_data, double *out_data, const double scalar, const int size);
__global__ void matrix_vector_multiplication(const GPUMatrix &A_sparse, const GPUMatrix &vector_dense, GPUMatrix result);



inline unsigned int div_up(unsigned int numerator, unsigned int denominator) { //numerator = zähler, denumerator = nenner
	unsigned int result = numerator / denominator;
	if (numerator % denominator) ++result;
	return result;
}



__global__ void sqaure_vector(const double *vector, double *result, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= size) { 
        return;
    } else {
        result[i] = vector[i] * vector[i];
    }

    __syncthreads();
}



double getNorm2(const GPUMatrix denseVector) {
    GPUMatrix tmp = matrix_alloc_gpu(denseVector.height, denseVector.width);
    double *result = new double[0];
    hipMalloc(&result, 1 * sizeof(double));

    int grids = div_up(denseVector.height, BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    int sh_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(double);
    
    sqaure_vector<<<grids, dimBlock>>>(denseVector.elements, tmp.elements, tmp.height * tmp.width); 
    norm2<<<grids, dimBlock, sh_memory_size>>>(tmp.elements, result);
    
    double r;
    hipMemcpy(&r, result, 1 * sizeof(double), hipMemcpyDeviceToHost);

    return sqrt(r);
}



__global__ void norm2(const double *in_data, double *result) {
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    sdata[tid] = in_data[i];        //load global data in sh_memory
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    //thread 0 writes in result back to global memory
    if (tid == 0) {
        result[0] = sdata[0];
    }
}




GPUMatrix get_add_subtract_vector(const GPUMatrix denseA, const GPUMatrix denseB, bool operation) {
    GPUMatrix result = matrix_alloc_gpu(denseA.height, denseA.width);

    int grids = div_up(denseA.height, BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    add_subtract_vector<<<grids, dimBlock>>>(denseA.elements, denseB.elements, result.elements, operation, denseA.width * denseA.height);

    return result;
}



__global__ void add_subtract_vector(const double *a, const double *b, double *c, const bool operation, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    //check if index out of range of vector
    if(i >= size) return;

    if(operation == true) {
        c[i] = a[i] + b[i];

    } else {
        c[i] = a[i] - b[i];
    }
    __syncthreads();
}



GPUMatrix multiply_scalar_vector(const GPUMatrix vector, const double scalar) {
    GPUMatrix result = matrix_alloc_gpu(vector.height, vector.width);

    int grids = div_up(vector.height, BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    scalar_vector<<<grids, dimBlock>>>(vector.elements, result.elements, scalar, vector.height * vector.width);
    
    return result;
}


__global__ void scalar_vector(const double *in_data, double *out_data, const double scalar, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        out_data[i] = scalar * in_data[i];
    }
    __syncthreads();
}




//shared memory
__global__ void matrix_vector_multiplication(const GPUMatrix &A_sparse, const GPUMatrix &vector_dense, GPUMatrix result) {
    //cuSparseCsrSpMV
}


GPUMatrix get_matrix_vector_multiplication(const GPUMatrix A_sparse, const GPUMatrix b_dense) {
    return b_dense;
}




GPUMatrix lsqr_algrithm(const GPUMatrix &A, const GPUMatrix &b, const double lambda, const double ebs) {
    GPUMatrix result = get_matrix_vector_multiplication(A, b);
    return result; 
}



CPUMatrix sparseLSQR_with_kernels(const CPUMatrix &A, const CPUMatrix &b, const double lambda, const double ebs) {
    CPUMatrix resultCPU = matrix_alloc_cpu(b.height, b.width);
    GPUMatrix resultGPU = matrix_alloc_gpu(b.height, b.width);

    GPUMatrix A_gpu = matrix_alloc_sparse_gpu(A.height, A.width, A.elementSize, A.rowSize, A.columnSize);
    GPUMatrix b_gpu = matrix_alloc_gpu(b.height, b.width);
    
    /* upload Matrix, vector */
    matrix_upload_cuSparse(A, A_gpu);
    matrix_upload(b, b_gpu);

    resultGPU = lsqr_algrithm(A_gpu, b_gpu, lambda, ebs);

    //printVector(b.height * b.width, resultGPU, "add vector");

    /* Download result */
    matrix_download(resultGPU, resultCPU);

    /* free GPU memory */
    hipFree(resultGPU.elements);
    hipFree(A_gpu.elements);
    hipFree(b_gpu.elements);

    return resultCPU;
}